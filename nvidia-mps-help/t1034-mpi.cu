#include "hip/hip_runtime.h"
/*
    Lifted from http://stackoverflow.com/questions/34709749/how-do-i-use-nvidia-multi-process-service-mps-to-run-multiple-non-mpi-cuda-app
*/



#include <stdio.h>
#include <stdlib.h>
#include <mpi.h>

#define MAX_DELAY 30

#define cudaCheckErrors(msg) \
  do { \
    hipError_t __err = hipGetLastError(); \
    if (__err != hipSuccess) { \
        fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
            msg, hipGetErrorString(__err), \
            __FILE__, __LINE__); \
        fprintf(stderr, "*** FAILED - ABORTING\n"); \
        exit(1); \
    } \
  } while (0)


#include <time.h>
#include <sys/time.h>
#define USECPSEC 1000000ULL

unsigned long long dtime_usec(unsigned long long start){

  timeval tv;
  gettimeofday(&tv, 0);
  return ((tv.tv_sec*USECPSEC)+tv.tv_usec)-start;
}

#define APPRX_CLKS_PER_SEC 1000000000ULL
__global__ void delay_kernel(unsigned seconds){

  unsigned long long dt = clock64();
  while (clock64() < (dt + (seconds*APPRX_CLKS_PER_SEC)));
}

int main(int argc, char *argv[]){
    MPI_Init(NULL, NULL);
    int world_size;
    MPI_Comm_size(MPI_COMM_WORLD, &world_size);
    int rank;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    char processor_name[MPI_MAX_PROCESSOR_NAME];
    int name_len;
    MPI_Get_processor_name(processor_name, &name_len);




  unsigned delay_t = 5; // seconds, approximately
  unsigned delay_t_r;
  if (argc > 1) delay_t_r = atoi(argv[1]);
  if ((delay_t_r > 0) && (delay_t_r < MAX_DELAY)) delay_t = delay_t_r;
  unsigned long long difft = dtime_usec(0);
  delay_kernel<<<1,1>>>(delay_t);
  hipDeviceSynchronize();
  cudaCheckErrors("kernel fail");
  difft = dtime_usec(difft);
  printf("kernel duration: %fs\n", difft/(float)USECPSEC);


  MPI_Finalize();

  return 0;
}

