/*
	Lifted from here https://developer.nvidia.com/cuda-education#HelloWorldSample
*/


#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void hello()
{
	printf("hello world, Im thread %d on block %d\n", threadIdx.x, blockIdx.x);
}


int main(void)
{
	int num_threads = 2;

	int num_blocks = 2;

	hello<<<num_blocks,num_threads>>>();
	hipDeviceSynchronize();

	return 0;
}




